#include "hip/hip_runtime.h"
#include <cstdlib>
#include <cstdio>
#include <cassert>

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>

#include <cute/tensor.hpp>
#include <cute/layout.hpp>

#include "cutlass/pipeline/sm90_pipeline.hpp"
#include "cutlass/gemm/collective/collective_builder.hpp"
#include <cute/algorithm/copy.hpp>
#include <cute/arch/copy_sm90.hpp>
#include <cute/atom/copy_traits.hpp>
#include <cute/arch/copy.hpp>
#include "cutlass/cutlass.h"
#include "cutlass/util/helper_cuda.hpp"
#include <cutlass/cluster_launch.hpp>
#include <cutlass/arch/barrier.h>
#include <cutlass/pipeline/pipeline.hpp>
#include <cutlass/arch/reg_reconfig.h>
#include "cutlass/tools/util/include/cutlass/util/print_error.hpp"

using namespace cute;

template <int STAGES, class TQ, class TK, class TV, class TO, 
         class SmemLayoutQ, class SmemLayoutK, class SmemLayoutV, class SmemLayoutO>
struct SharedStorage {
    cute::array_aligned<TQ, cute::cosize_v<SmemLayoutQ>> smem_q;
    cute::array_aligned<TK, cute::cosize_v<SmemLayoutK>> smem_k;
    union {
        cute::array_aligned<TV, cute::cosize_v<SmemLayoutV>> smem_v;
        cute::array_aligned<TO, cute::cosize_v<SmemLayoutO>> smem_o;
    };
    struct {
        cutlass::arch::ClusterTransactionBarrier barrier_Q;
        cutlass::arch::ClusterBarrier barrier_O;
        typename cutlass::PipelineTmaAsync<STAGES>::SharedStorage pipeline_k;
        typename cutlass::PipelineTmaAsync<STAGES>::SharedStorage pipeline_v;
    };
};

// Device kernel template
template <typename TQ, typename TK, typename TV, typename TO, 
            class CtaTiler,
            class SmemLayoutQ, class TmaQ,
            class SmemLayoutK, class TmaK,
            class SmemLayoutV, class TmaV,
            class SmemLayoutO, 
            class TiledMmaQK, class TiledMmaPV,
            int kWarps, int kWarpGroups, int kConsumerWGs, int kThreads,
            int bM, int bN, int bK, int PIPE, int cluster_M>
__global__ static
__launch_bounds__(kThreads, 1)
void
flash_attn_device(CtaTiler cta_tiler, 
                    int num_blocks_m, int num_blocks_n,
                    TQ const* Q, CUTLASS_GRID_CONSTANT TmaQ const tma_q,
                    TK const* K, CUTLASS_GRID_CONSTANT TmaK const tma_k,
                    TV const* V, CUTLASS_GRID_CONSTANT TmaV const tma_v,
                    TO* O,
                    TiledMmaQK mma_qk, TiledMmaPV mma_pv)
{
    using MainloopPipeline = cutlass::PipelineTmaAsync<PIPE>;
    using PipelineState = typename MainloopPipeline::PipelineState;
    typename MainloopPipeline::Params params;

    extern __shared__ char shared_memory[];
    using SharedStorage = SharedStorage<PIPE, TQ, TK, TV, TO, SmemLayoutQ, SmemLayoutK, SmemLayoutV, SmemLayoutO>;
    SharedStorage &smem = *reinterpret_cast<SharedStorage*>(shared_memory);

    const int warp_idx = cutlass::canonical_warp_idx_sync();
    const int warp_group_idx = cutlass::canonical_warp_group_idx();
    const int lane_predicate = cute::elect_one_sync();

    if (warp_idx == 0 && lane_predicate) {
        prefetch_tma_descriptor(tma_q.get_tma_descriptor());
        prefetch_tma_descriptor(tma_k.get_tma_descriptor());
        prefetch_tma_descriptor(tma_v.get_tma_descriptor());
    }

    params.is_leader = threadIdx.x % 128 == 0;
    params.num_consumers = kConsumerWGs * 128;

    static constexpr uint32_t TmaTransactionBytesQ = static_cast<uint32_t>(size(SmemLayoutQ{}) * cutlass::sizeof_bits_v<TQ> / 8);
    static constexpr uint32_t TmaTransactionBytesk = static_cast<uint32_t>(size(take<0,2>(SmemLayoutK{})) * cutlass::sizeof_bits_v<TQ> / 8); // faster than init a new aligned_array or array_engine
    static constexpr uint32_t TmaTransactionBytesV = static_cast<uint32_t>(size(take<0,2>(SmemLayoutV{})) * cutlass::sizeof_bits_v<TQ> / 8); 
}

template <class TQ, class TK, class TV, class TO, int D>
void run_flash_attn(int B, int T, int NH,
                    TQ const* Q, 
                    TK const* K, 
                    TV const* V, 
                    TO const* O,
                    hipStream_t stream = 0) 
{
    auto batch_size = int(B);
    auto seq_len = int(T);
    auto n_heads = int(NH);

    static constexpr int bM = 256;
    static constexpr int bN = 128;
    static constexpr int HEAD_DIM = D;
    static constexpr int bP = 3;
    static constexpr int CLUSER_M = 1;

    static constexpr int q_head_dim_stride = 1;  // stride-1 in head dimension
    static constexpr int q_seq_stride = HEAD_DIM;
    int q_head_stride = HEAD_DIM * seq_len;
    int q_batch_stride = HEAD_DIM * seq_len * n_heads;

    static constexpr int k_head_dim_stride = 1;  // stride-1 in head dimension
    static constexpr int k_seq_stride = HEAD_DIM;
    int k_head_stride = HEAD_DIM * seq_len;
    int k_batch_stride = HEAD_DIM * seq_len * n_heads;

    static constexpr int v_head_dim_stride = 1;  // stride-1 in head dimension
    static constexpr int v_seq_stride = HEAD_DIM;
    int v_head_stride = HEAD_DIM * seq_len;
    int v_batch_stride = HEAD_DIM * seq_len * n_heads;
    
    static constexpr int o_head_dim_stride = 1;  // stride-1 in head dimension
    static constexpr int o_seq_stride = HEAD_DIM;
    int o_head_stride = HEAD_DIM * seq_len;
    int o_batch_stride = HEAD_DIM * seq_len * n_heads;

    using TiledShape_MNK = Shape<Int<bM>, Int<bN>, Int<HEAD_DIM>>;

    auto SmemLayoutAtomQ = cutlass::gemm::collective::detail::ss_smem_selector<GMMA::Major::K, TQ, Int<bM>, Int<HEAD_DIM>>();
    auto SmemLayoutAtomK = cutlass::gemm::collective::detail::ss_smem_selector<GMMA::Major::K, TK, Int<bN>, Int<HEAD_DIM>>();
    auto SmemLayoutAtomV = cutlass::gemm::collective::detail::ss_smem_selector<GMMA::Major::K, TV, Int<bN>, Int<HEAD_DIM>>();
    auto SmemLayoutAtomO = cutlass::gemm::collective::detail::ss_smem_selector<GMMA::Major::K, TO, Int<bM>, Int<HEAD_DIM>>();

    auto SmemLayoutQ = tile_to_shape(SmemLayoutAtomQ, make_shape(Int<bM>{}, Int<HEAD_DIM>{}));
    auto SmemLayoutK = tile_to_shape(SmemLayoutAtomK, make_shape(Int<bN>{}, Int<HEAD_DIM>{}, Int<bP>{}));
    auto SmemLayoutO = tile_to_shape(SmemLayoutAtomQ, make_shape(Int<bM>{}, Int<HEAD_DIM>{}));

    auto SmemLayoutV = SmemLayoutK;
    // auto SmemLayoutVt = cute::composition(SmemLayoutV, make_layout(make_shape(Int<HEAD_DIM>{}, Int<bN>{}, Int<bP>{}), make_stride(Int<bN>{}, _1{}, Int<size(SmemLayoutV(_,_,_,_0{})>))));
    auto SmemLayoutVt = cute::composition(SmemLayoutV, make_ordered_layout(make_shape(Int<HEAD_DIM>{}, Int<bN>{}, Int<bP>{}), Step<_2, _1, _3>{}));

    auto LayoutQ = make_layout(
        make_shape(seq_len, HEAD_DIM, n_heads, batch_size),
        make_stride(q_seq_stride, q_head_dim_stride, q_head_stride, q_batch_stride)                                   // stride for B (batch_size)
    );

    auto LayoutK = make_layout(
        make_shape(seq_len, HEAD_DIM, n_heads, batch_size),
        make_stride(k_seq_stride, k_head_dim_stride, k_head_stride, k_batch_stride)                                   // stride for B (batch_size)
    );

    auto LayoutV = make_layout(
        make_shape(seq_len, HEAD_DIM, n_heads, batch_size),
        make_stride(v_seq_stride, v_head_dim_stride, v_head_stride, v_batch_stride)                                   // stride for B (batch_size)
    );

    Tensor mQ = make_tensor(make_gmem_ptr(Q), LayoutQ);
    Tensor mK = make_tensor(make_gmem_ptr(K), LayoutK);
    Tensor mV = make_tensor(make_gmem_ptr(V), LayoutV);

    auto tma_q = cute::make_tma_copy(
        SM90_TMA_LOAD{},
        mQ,
        SmemLayoutQ,
        make_shape(Int<bM>{}, Int<HEAD_DIM>{}),
        _1{}
    );

    auto tma_k = cute::make_tma_copy(
        SM90_TMA_LOAD{},
        mK,
        SmemLayoutK(_,_,_0{}),
        make_shape(Int<bN>{}, Int<HEAD_DIM>{}),
        _1{}
    );

    auto tma_v = cute::make_tma_copy(
        SM90_TMA_LOAD{},
        mV,
        SmemLayoutV(_,_,_0{}),
        make_shape(Int<bN>{}, Int<HEAD_DIM>{}),
        _1{}
    );

    using AtomLayoutMNK = Layout<Shape<Int<bM / 64>, _1, _1>>;    

    TiledMMA TiledMmaQK = make_tiled_mma(GMMA::ss_op_selector<TQ, TK, float, TiledShape_MNK>(), AtomLayoutMNK{});
    TiledMMA TiledMmaPV = make_tiled_mma(GMMA::ss_op_selector<TK, TV, float, TiledShape_MNK, GMMA::Major::K, GMMA::Major::MN>(), 
                                        AtomLayoutMNK{});
    
    static constexpr int NUM_WARPS = ((bM / 64) + 1) * 4;
    static constexpr int NUM_THREADS = NUM_WARPS * 32;
    static constexpr int NUM_WARPGROUPS = NUM_WARPS / 4;
    static constexpr int NUM_CONSUMER_GROUPS = NUM_WARPGROUPS - 1;
    static constexpr int NUM_PRODUCER_GROUPS = 1;
    static constexpr int NUM_CONSUMER_THREADS = NUM_CONSUMER_GROUPS * cutlass::NumThreadsPerWarp;
    static constexpr int NUM_PRODUCER_THREADS = cutlass::NumThreadsPerWarp;
    static constexpr int NUM_MMA_THREADS = size(TiledMmaQK);

    int num_tiles_q = cutlass::ceil_div(seq_len, bM);
    void const* kernel = reinterpret_cast<void const*>(
        &flash_attn_device<TQ, TK, TV, TO, 
                          decltype(TiledShape_MNK{}),
                          decltype(SmemLayoutQ), decltype(tma_q),
                          decltype(SmemLayoutK), decltype(tma_k),
                          decltype(SmemLayoutV), decltype(tma_v),
                          decltype(SmemLayoutO), 
                          decltype(TiledMmaQK), decltype(TiledMmaPV),
                          NUM_WARPS, NUM_WARPGROUPS, NUM_CONSUMER_GROUPS, NUM_THREADS,
                          bM, bN, HEAD_DIM, bP, CLUSER_M>);

    dim3 dimBlock(NUM_THREADS);
    dim3 dimCluster(CLUSER_M, 1, 1);
    dim3 dimGrid(num_tiles_q, n_heads * batch_size);
    
    int smem_bytes = int(sizeof(SharedStorage<bP, TQ, TK, TV, TO,
                                            decltype(SmemLayoutQ), decltype(SmemLayoutK),
                                            decltype(SmemLayoutV), decltype(SmemLayoutO)>));
    
    CUTE_CHECK_ERROR(hipFuncSetAttribute(reinterpret_cast<const void*>(kernel),
        hipFuncAttributeMaxDynamicSharedMemorySize,
        smem_bytes));

    cutlass::ClusterLaunchParams params = {dimGrid, dimBlock, dimCluster, smem_bytes};
    cutlass::Status status = cutlass::launch_kernel_on_cluster(params, kernel, 
                                                            TiledShape_MNK{},
                                                            num_tiles_q, num_tiles_q,
                                                            Q, tma_q,
                                                            K, tma_k, 
                                                            V, tma_v,
                                                            O, 
                                                            TiledMmaQK, TiledMmaPV);
}

template <class TQ, class TK, class TV, class TO>
void flash_attention(int B, int T, int NH, int D,
    TQ const* Q,
    TK const* K,
    TV const* V,
    TO const* O,
    hipStream_t stream = 0)
{
    run_flash_attn<TQ, TK, TV, TO, 128>(B, T, NH, Q, K, V, O, stream);
}



int main(int argc, char** argv) {
    int B = 8;
    int T = 2048;      // sequence length
    int NH = 8;      // number of heads
    int HEAD_DIM = 128;     // head size
    int D = NH * HEAD_DIM; // model dim per token

    using TQ = cute::bfloat16_t;
    using TK = cute::bfloat16_t;
    using TV = cute::bfloat16_t;
    using TO = cute::bfloat16_t;

    // Replace lines 141-144 with:
    thrust::host_vector<TQ> h_Q(B*T*D);
    thrust::host_vector<TK> h_K(B*T*D);
    thrust::host_vector<TV> h_V(B*T*D);
    thrust::host_vector<TO> h_O(B*T*D);

    for (int j = 0; j < B*T*D; ++j) h_Q[j] = TQ(int((rand() % 2) ? 1 : -1));
    for (int j = 0; j < B*T*D; ++j) h_K[j] = TK(int((rand() % 2) ? 1 : -1));
    for (int j = 0; j < B*T*D; ++j) h_V[j] = TV(int((rand() % 2) ? 1 : -1));
    for (int j = 0; j < B*T*D; ++j) h_O[j] = TV(0);

    thrust::device_vector<TQ> d_Q = h_Q;
    thrust::device_vector<TK> d_K = h_K;
    thrust::device_vector<TV> d_V = h_V;
    thrust::device_vector<TV> d_O = h_O;
    thrust::device_vector<TO> d_O_ref = h_O;  // Reference result

    // // Initialize cuBLAS
    // hipblasHandle_t cublas_handle;
    // hipblasCreate(&cublas_handle);

    // printf("Running correctness verification...\n");
    
    // // Run cuBLAS reference
    // d_C_ref = h_C;  // Reset
    // run_cublas_gemm(cublas_handle, transA, transB, m, n, k, 
    //                static_cast<float>(alpha),
    //                d_A.data().get(), ldA,
    //                d_B.data().get(), ldB,
    //                static_cast<float>(beta),
    //                d_C_ref.data().get(), ldC);
    
    // Run CuTe implementation
    d_O = h_O;  // Reset
    flash_attention<TQ, TK, TV, TO>(B, T, NH, HEAD_DIM,
        d_Q.data().get(),
        d_K.data().get(),
        d_V.data().get(),
        d_O.data().get());
    
    // Copy results back to host for verification
    thrust::host_vector<TO> cute_result = d_O;
    thrust::host_vector<TO> cudnn_result = d_O_ref;
    
    // Verify correctness
    // bool passed = verify_matrix(cudnn_result, cute_result, B, T, D);
    // if (passed) {
    //     printf("✓ Correctness verification passed!\n");
    // } else {
    //     printf("✗ Correctness verification FAILED!\n");
    //     hipblasDestroy(cublas_handle);
    //     return 1;
    // }
}
